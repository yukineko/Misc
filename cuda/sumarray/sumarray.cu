
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

void sumArraysOnHost(float* A, float* B, float* C, const int N){
    int idx;
    for(idx = 0; idx < N; idx++){
        C[idx] = A[idx] + B[idx];
    }
}

void initialData(float* ip, int size){
    time_t t;
    srand((unsigned int) time(&t));

    for(int i = 0; i < size; i++){
        ip[i] = (float)(rand() & 0xff) / 10.0f;
    }
}

    
int main(){
    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *h_C;
    h_A = (float*) malloc(nBytes);
    h_B = (float*) malloc(nBytes);
    h_C = (float*) malloc(nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, nBytes);
    hipMalloc(&d_B, nBytes);
    hipMalloc(&d_C, nBytes);
    
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, nBytes, hipMemcpyHostToDevice);
    
    sumArraysOnHost(h_A, h_B, h_C, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
