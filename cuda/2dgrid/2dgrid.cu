#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sample-common.h"

void sumMatrixOnHost(float* a, float* b, float* host, int x, int y){
    printf("matrix\n");
    float* ia = a;
    float* ib = b;
    float* ic = host;
    for(int i = 0;i < x; i++){
        for(int j = 0; j < y; j++){
            ic[j] = ia[j] + ib[j];
        }
        ia += x;
        ib += x;
        ic += x;
    }
}

__global__ void sumMatrixOnGpu(float* a, float* b, float* gpu, int x, int y){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * ix + x;
    if(ix < x && iy < y){
        gpu[idx] = a[idx] + b[idx];
    }
}


__global__ void sumMatrixUSING1D(float* a, float* b, float* gpu, int x, int y){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if(ix < x){
        for(int iy = 0; iy < y; iy++){
            int idx = iy * ix + x;
            gpu[idx] = a[idx] + b[idx];
        }
    }
}
int main(int argc, char* argv[]){
    printf("%s starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, dev));
    printf("Using Device %d\n", dev);

    
    CHECK(hipSetDevice(dev));
    int nx = 1 << 13;
    int ny = 1 << 13;
    int nxy = nx * ny;
    size_t nBytes = nxy * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    double iStart, iElasp;
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    clearData(hostRef, nxy);
    clearData(gpuRef, nxy);
    
    iStart = cpuSecond();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElasp = cpuSecond() - iStart;
    printf("sumMatrixOnHost %d %d  time=%f\n", nx, ny, iElasp);


    int dimx = 32, dimy = 16;

    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void**)&d_A, nBytes));
    CHECK(hipMalloc((void**)&d_B, nBytes));
    CHECK(hipMalloc((void**)&d_C, nBytes));
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    iStart = cpuSecond();    
    sumMatrixOnGpu<<<grid, block>>>(d_A, d_B, d_C, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElasp = cpuSecond() - iStart;    
    printf("sumMatrixOnGpu<<<(%d, %d)(%d, %d)>>> %d, %d  time=%f\n",
           grid.x, grid.y,
           block.x, block.y,
           nx, ny,
           iElasp);
              
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    int ret = checkResult(hostRef, gpuRef, nxy);
    if(ret != 0){
        fprintf(stderr, "check error!!\n");
        exit(-1);
    }
    
    iStart = cpuSecond();    
    dim3 block2(32, 1);
    dim3 grid2((nx + block.x - 1) / block.x, 1);
    sumMatrixUSING1D<<<grid2, block2>>>(d_A, d_B, d_C, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElasp = cpuSecond() - iStart;    
    printf("sumMatrixUSING 1D<<<(%d, %d)(%d, %d)>>> %d, %d  time=%f\n",
           grid2.x, grid2.y,
           block2.x, block2.y,
           nx, ny,
           iElasp);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    ret = checkResult(hostRef, gpuRef, nxy);
    if(ret != 0){
        fprintf(stderr, "check error!!\n");
        exit(-1);
    }
    
    
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);    
    return 0;
}

