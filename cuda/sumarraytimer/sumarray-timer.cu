#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#include "sample-common.h"
#define LENGTH 1024


void sumArrayOnHost(float* A, float* B, float* C, int size){
    for(int idx = 0; idx < size; idx++){
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArrayOnGpu(float* A, float* B, float* C, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("blockIdx.x = %d, blockDim.x = %d, threadIdx.x = %d, value = %d\n",
    //        blockIdx.x,
    //        blockDim.x,
    //        threadIdx.x,
    //        i
    //        );
    if(i < size){
       C[i] = A[i] + B[i];
    }
}

void testAtHost(float* h_A, float* h_B, float* hostRef, float* gpuRef, int size){
    double iStart, iElaps;    
    iStart = cpuSecond();
    sumArrayOnHost(h_A, h_B, hostRef, size);
    iElaps = cpuSecond() - iStart;    
    printf("sumArrayOnHost Time elapsed %f sec\n", iElaps);
}

void testAtGpu(float* h_A, float* h_B, float* gpuRef, int size){
    float *d_A, *d_B, *d_C;
    int nBytes = size * sizeof(float);
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));
    
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
    
    double iStart, iElasp;
    
    dim3 block(LENGTH);
    dim3 grid((size + block.x -1) / block.x);
    iStart = cpuSecond();    
    sumArrayOnGpu<<<grid, block>>>(d_A, d_B, d_C, size);

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    iElasp = cpuSecond() - iStart;
    printf("<<<grid %d, block %d>>> %f sec\n", grid.x, block.x, iElasp);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
}
    

int checkResult(float* A, float* B, int size){
    for(int i = 0;i < size; i++){
        if(A[i] != B[i]){
            return i;
        }
    }
    return 0;
}


int main(int argc, char* argv[]){
    printf("%s starting ..\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;

    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s,\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
    int nElem = 1 << 24;
    
    printf("Vector size %d\n", nElem);
    float *h_A, *h_B, *gpuRef, *hostRef;
    //    init(h_A, h_B, gpuRef, hostRef, nElem);
    size_t nBytes = nElem * sizeof(float);
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);
    clearData(hostRef, nElem);
    clearData(gpuRef, nElem);
    
    testAtGpu(h_A, h_B, gpuRef, nElem);
    testAtHost(h_A, h_B, hostRef, gpuRef, nElem);    
    int nRet = checkResult(hostRef, gpuRef, nElem);
    if( nRet != 0 ){
        printf("check result error %d\n", nRet);
    }
    
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    
    return 0;
}
