#include "hip/hip_runtime.h"
#include "sample-common.h"
#include <stdio.h>

__global__ void printMatrixIndex(int* mat, int nx, int ny)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int idx = iy * nx + ix;
    printf("blockIdx(%d, %d), blockDim(%d, %d), threadIdx(%d, %d), idx = %d, val = %d\n",
           blockIdx.x, blockIdx.y,
           blockDim.x, blockDim.y,
           threadIdx.x, threadIdx.y,
           idx,
           mat[idx]
           );
    
}

int main(int argc, char* argv[]){
    printf("%s Starting...\n", argv[0]);
    int dev = 0;
    cudaDeviceInit(dev);

    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;

    int nBytes = nxy * sizeof(int);
    int *h_A;
    h_A = (int*)malloc(nBytes);

    initDataSeq(h_A, nxy);

    printMatrixInt(h_A, nx, ny);

    int* d_Mat;
    CHECK(hipMalloc((void**)&d_Mat, nBytes));
    CHECK(hipMemcpy(d_Mat, h_A, nBytes, hipMemcpyHostToDevice));

    dim3 block(4,2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    printMatrixIndex<<<grid, block>>>(d_Mat, nx, ny);

    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(d_Mat));
    free(h_A);

    CHECK(hipDeviceReset());
    return 0;
}