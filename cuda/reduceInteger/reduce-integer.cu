#include "hip/hip_runtime.h"
#include "sample-common.h"
#include <stdio.h>
#include <stdbool.h>
#include <string.h>

int recursiveReduce(int* data, size_t size){
    if(size == 1){
        return data[0];
    }

    size_t stride = size / 2;

    for(int i = 0; i < stride; i++){
        data[i] += data[i + stride];
    }
    return recursiveReduce(data, stride);
}

int testCpuRecursiveReduce(int* tmp, size_t size){
    double iStart, iElasp;
    
    iStart = cpuSecond();
    int cpu_sum = recursiveReduce(tmp, size);
    iElasp = cpuSecond() - iStart;
    printf("cpu reduce elapsed %f sec cpu_sum %d\n", iElasp, cpu_sum);
    return cpu_sum;
    
}

__global__ void reduceNeiborbored(int *g_idata, int *g_odata, unsigned int size){
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int* idata = g_idata + blockIdx.x * blockDim.x;
    if(idx >= size){
        return;
    }

    for(int stride = 1; stride < blockDim.x; stride *= 2){
        if((tid % (2 * stride)) == 0){
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();        
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}

int testGpuReduceNeiborbored(int* data, size_t size){
    double iStart, iElasp;

    int *d_idata, *d_odata;
    int blocksize = 512;

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    
    size_t bytes = size * sizeof(int);
    size_t gridbytes = grid.x * sizeof(int);
    
    // malloc result
    int* result = (int*) malloc(gridbytes);
    printf("grid %d block %d\n", grid.x, block.x);
    
    CHECK(hipMalloc((void**)&d_idata, bytes));
    CHECK(hipMalloc((void**)&d_odata, gridbytes));
    CHECK(hipMemcpy(d_idata, data, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());    
    iStart = cpuSecond();
    reduceNeiborbored<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(result, d_odata, gridbytes, hipMemcpyDeviceToHost));
    int gpu_sum = 0;
    for(int i = 0; i < grid.x; i++){
        gpu_sum += result[i];
    }
    // free result
    free(result);
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));    
    
    iElasp = cpuSecond() - iStart;    
    printf("gpu Neiborbored elapsed %f sec, gpu_sum = %d, <<< grid %d, block %d >>>\n", iElasp, gpu_sum, grid.x, block.x);
    return 0;
}

__global__ void reduceNeiborless(int* g_idata, int* g_odata, size_t size){
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int *idata = g_idata + blockIdx.x * blockDim.x;
    if(idx >= size){
        return;
    }

    for(int stride = 1; stride < blockDim.x; stride *= 2){
        int index = 2 * stride * tid;
        if(index < blockDim.x){
            idata[index] += idata[index + stride];
        }
        __syncthreads();
    }
    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}


int testGpuReduceNeiborless(int* data, unsigned int size){
    double iStart, iElasp;

    int *d_idata, *d_odata;
    int blocksize = 512;

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    
    size_t bytes = size * sizeof(int);
    size_t gridbytes = grid.x * sizeof(int);
    
    // malloc result
    int* result = (int*) malloc(gridbytes);
    printf("grid %d block %d\n", grid.x, block.x);
    
    CHECK(hipMalloc((void**)&d_idata, bytes));
    CHECK(hipMalloc((void**)&d_odata, gridbytes));
    CHECK(hipMemcpy(d_idata, data, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());    
    iStart = cpuSecond();
    reduceNeiborless<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(result, d_odata, gridbytes, hipMemcpyDeviceToHost));
    int gpu_sum = 0;
    for(int i = 0; i < grid.x; i++){
        gpu_sum += result[i];
    }
    // free result
    free(result);
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));    
    
    iElasp = cpuSecond() - iStart;    
    printf("gpu Neiborless elapsed %f sec, gpu_sum = %d, <<< grid %d, block %d >>>\n", iElasp, gpu_sum, grid.x, block.x);
    return gpu_sum;
}


__global__ void reduceInterleave(int* g_idata, int* g_odata, size_t size){
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int *idata = g_idata + blockIdx.x * blockDim.x;
    if(idx >= size){
        return;
    }

    for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
        if(tid < stride){
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}


int testGpuReduceInterleave(int* data, size_t size){
    double iStart, iElasp;

    int *d_idata, *d_odata;
    int blocksize = 512;

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    
    size_t bytes = size * sizeof(int);
    size_t gridbytes = grid.x * sizeof(int);
    
    // malloc result
    int* result = (int*) malloc(gridbytes);
    printf("grid %d block %d\n", grid.x, block.x);
    
    CHECK(hipMalloc((void**)&d_idata, bytes));
    CHECK(hipMalloc((void**)&d_odata, gridbytes));
    CHECK(hipMemcpy(d_idata, data, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());    
    iStart = cpuSecond();
    reduceInterleave<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(result, d_odata, gridbytes, hipMemcpyDeviceToHost));
    int gpu_sum = 0;
    for(int i = 0; i < grid.x; i++){
        gpu_sum += result[i];
    }
    // free result
    free(result);
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));    
    
    iElasp = cpuSecond() - iStart;    
    printf("gpu Interleave elapsed %f sec, gpu_sum = %d, <<< grid %d, block %d >>>\n", iElasp, gpu_sum, grid.x, block.x);
    return gpu_sum;    
}


__global__ void reduceUnrolling2(int* g_idata, int* g_odata, size_t size){
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;
    if(idx + blockDim.x < size){
        g_idata[idx] += g_idata[idx + blockDim.x];
    }
    __syncthreads();

    for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
        if(tid < stride){
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}

int testGpuReduceUnrolling2(int* data, size_t size){
    double iStart, iElasp;

    int *d_idata, *d_odata;
    int blocksize = 512;

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    
    size_t bytes = size * sizeof(int);
    size_t gridbytes = grid.x * sizeof(int);
    
    // malloc result
    int* result = (int*) malloc(gridbytes);
    printf("grid %d block %d\n", grid.x, block.x);
    
    CHECK(hipMalloc((void**)&d_idata, bytes));
    CHECK(hipMalloc((void**)&d_odata, gridbytes));
    CHECK(hipMemcpy(d_idata, data, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());    
    iStart = cpuSecond();
    reduceUnrolling2<<<grid.x / 2, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(result, d_odata, gridbytes / 2, hipMemcpyDeviceToHost));
    int gpu_sum = 0;
    for(int i = 0; i < grid.x / 2; i++){
        gpu_sum += result[i];
    }
    // free result
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));    
    free(result);
    
    iElasp = cpuSecond() - iStart;    
    printf("gpu Unroll 2 elapsed %f sec, gpu_sum = %d, <<< grid %d, block %d >>>\n", iElasp, gpu_sum, grid.x / 2, block.x);
    return gpu_sum;    
}


__global__ void reduceUnrolling8(int* g_idata, int* g_odata, size_t size){
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;
    if(idx + 7 * blockDim.x < size){
        int a0 = g_idata[idx];
        int a1 = g_idata[idx + blockDim.x];
        int a2 = g_idata[idx + 2 * blockDim.x];
        int a3 = g_idata[idx + 3 * blockDim.x];
        int b0 = g_idata[idx + 4 * blockDim.x];
        int b1 = g_idata[idx + 5 * blockDim.x];
        int b2 = g_idata[idx + 6 * blockDim.x];
        int b3 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a0 + a1 + a2 + a3 + b0 + b1 + b2 + b3;
    }
    __syncthreads();

    for(int stride = blockDim.x / 2; stride > 32; stride >>= 1){
        if(tid < stride){
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    if(tid < 32){
        volatile int * vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
}

int testGpuReduceUnrolling8(int* data, size_t size){
    double iStart, iElasp;

    int *d_idata, *d_odata;
    int blocksize = 512;

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    
    size_t bytes = size * sizeof(int);
    size_t gridbytes = grid.x * sizeof(int);
    
    // malloc result
    int* result = (int*) malloc(gridbytes);
    printf("grid %d block %d\n", grid.x, block.x);
    
    CHECK(hipMalloc((void**)&d_idata, bytes));
    CHECK(hipMalloc((void**)&d_odata, gridbytes));
    CHECK(hipMemcpy(d_idata, data, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());    
    iStart = cpuSecond();
    reduceUnrolling8<<<grid.x / 8, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(result, d_odata, gridbytes / 8, hipMemcpyDeviceToHost));
    int gpu_sum = 0;
    for(int i = 0; i < grid.x / 8; i++){
        gpu_sum += result[i];
    }
    // free result
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));    
    free(result);
    
    iElasp = cpuSecond() - iStart;    
    printf("gpu Unroll 8 elapsed %f sec, gpu_sum = %d, <<< grid %d, block %d >>>\n", iElasp, gpu_sum, grid.x / 8, block.x);
    return gpu_sum;    
}

__global__ void reduceCompleteUnroll(int* g_idata, int* g_odata, size_t size){
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;
    if(idx + 7 * blockDim.x < size){
        int a0 = g_idata[idx];
        int a1 = g_idata[idx + blockDim.x];
        int a2 = g_idata[idx + 2 * blockDim.x];
        int a3 = g_idata[idx + 3 * blockDim.x];
        int b0 = g_idata[idx + 4 * blockDim.x];
        int b1 = g_idata[idx + 5 * blockDim.x];
        int b2 = g_idata[idx + 6 * blockDim.x];
        int b3 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a0 + a1 + a2 + a3 + b0 + b1 + b2 + b3;
    }
    __syncthreads();
    if(blockDim.x >= 1024 && tid < 512){
        idata[tid] += idata[tid + 512];
        __syncthreads();
    }
    if(blockDim.x >= 512 && tid < 256){
        idata[tid] += idata[tid + 256];
        __syncthreads();
    }
    if(blockDim.x >= 256 && tid < 128){
        idata[tid] += idata[tid + 128];
        __syncthreads();
    }

    if(blockDim.x >= 128 && tid < 64){
        idata[tid] += idata[tid + 64];
        __syncthreads();
    }

    if(tid < 32){
        volatile int * vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    if(tid == 0){
        g_odata[blockIdx.x] = idata[0];
    }
   
}
int testReduceCompleteUnroll(int* data, size_t size){
    double iStart, iElasp;

    int *d_idata, *d_odata;
    int blocksize = 512;

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    
    size_t bytes = size * sizeof(int);
    size_t gridbytes = grid.x * sizeof(int);
    
    // malloc result
    int* result = (int*) malloc(gridbytes);
    printf("grid %d block %d\n", grid.x, block.x);
    
    CHECK(hipMalloc((void**)&d_idata, bytes));
    CHECK(hipMalloc((void**)&d_odata, gridbytes));
    CHECK(hipMemcpy(d_idata, data, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());    
    iStart = cpuSecond();
    reduceUnrolling8<<<grid.x / 8, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(result, d_odata, gridbytes / 8, hipMemcpyDeviceToHost));
    int gpu_sum = 0;
    for(int i = 0; i < grid.x / 8; i++){
        gpu_sum += result[i];
    }
    // free result
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));    
    free(result);
    
    iElasp = cpuSecond() - iStart;    
    printf("gpu Unroll Complete elapsed %f sec, gpu_sum = %d, <<< grid %d, block %d >>>\n", iElasp, gpu_sum, grid.x / 8, block.x);
    return gpu_sum;        
}


int main(int argc, char* argv[]){
    int dev = 0;
    printf("%s starting.\n", argv[0]);
    hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, dev));

    printf("device %d, %s\n", dev, prop.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;
    int size = 1 << 24;
    printf("with arrray size = %d\n", size);

    int blocksize = 512;
    if(argc > 1){
        blocksize = atoi(argv[1]);
    }

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);

    size_t bytes = size * sizeof(int);
    size_t gridbytes = grid.x * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(gridbytes);
    int *tmp = (int *)malloc(bytes);

    initialDataInt(h_idata, size);
    
    memcpy(tmp, h_idata, bytes);
    int result1 = testGpuReduceNeiborbored(tmp, size);
    
    memcpy(tmp, h_idata, bytes);        
    int result2 = testCpuRecursiveReduce(tmp, size);

    memcpy(tmp, h_idata, bytes);
    int result3 = testGpuReduceNeiborless(tmp, size);

    memcpy(tmp, h_idata, bytes);
    int result4 = testGpuReduceInterleave(tmp, size);

    memcpy(tmp, h_idata, bytes);
    int result5 = testGpuReduceUnrolling2(tmp, size);

    memcpy(tmp, h_idata, bytes);
    int result6 = testGpuReduceUnrolling8(tmp, size);

    memcpy(tmp, h_idata, bytes);    
    int result7 = testReduceCompleteUnroll(tmp, size);
    return 0;
}