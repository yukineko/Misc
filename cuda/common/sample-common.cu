#include "sample-common.h"
#include <stdio.h>
#include <sys/time.h>

double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6 );
}

void initialData(float* data, int size){
    time_t t;
    srand((unsigned) time(&t));

    for(int i = 0; i < size; i++){
        data[i] = (float)(rand() & 0xff) / 10.0f;
    }
}

void clearData(float* data, int size){
    for(int i = 0; i < size; i++){
        data[i] = 0;
    }    
}

void initAtNum(int* A, size_t size, int val)
{
    for(int i = 0;i < size; i++){
        A[i] = val;
    }
}

void initDataSeq(int* A, size_t size)
{
    for(int i = 0;i < size; i++){
        A[i] = i;
    }
}

void printMatrixInt(int* A, const int x, const int y){
    int *ic = A;
    printf("Matrix: (%d, %d)\n", x, y);
    for(int iy = 0; iy < y; iy++){
        for(int ix = 0; ix < x; ix++){
            printf("%3d, ", ic[ix]);
        }
        ic += x;
        printf("\n");
    }
    printf("\n");
}

void cudaDeviceInit(int dev){
    hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, dev));
    printf("Using device %d, %s\n", dev, prop.name);
    CHECK(hipSetDevice(dev));
}
