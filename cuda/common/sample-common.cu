#include "sample-common.h"
#include <stdio.h>
#include <sys/time.h>

double cpuSecond(){
    struct timespec ts;
    clock_gettime(CLOCK_REALTIME, &ts);
    double sec = ts.tv_sec;
    double usec = (double)ts.tv_nsec / (double)(1000 * 1000 * 1000);
    //    printf("sec = %f\n", sec);
    //    printf("usec = %f\n", usec);
    return sec + usec;
}

void initialData(float* data, int size){
    time_t t;
    srand((unsigned) time(&t));

    for(int i = 0; i < size; i++){
        data[i] = (float)(rand() & 0xff) / 10.0f;
    }
}

void initialDataInt(int* data, int size){
    time_t t;
    srand((unsigned) time(&t));

    for(int i = 0; i < size; i++){
        data[i] = (int)(rand() & 0xff);
    }
}

void clearData(float* data, int size){
    for(int i = 0; i < size; i++){
        data[i] = 0;
    }    
}

void initAtNum(int* A, size_t size, int val)
{
    for(int i = 0;i < size; i++){
        A[i] = val;
    }
}

void initDataSeq(int* A, size_t size)
{
    for(int i = 0;i < size; i++){
        A[i] = i;
    }
}

void printMatrixInt(int* A, const int x, const int y){
    int *ic = A;
    printf("Matrix: (%d, %d)\n", x, y);
    for(int iy = 0; iy < y; iy++){
        for(int ix = 0; ix < x; ix++){
            printf("%3d, ", ic[ix]);
        }
        ic += x;
        printf("\n");
    }
    printf("\n");
}

void cudaDeviceInit(int dev){
    hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, dev));
    printf("Using device %d, %s\n", dev, prop.name);
    CHECK(hipSetDevice(dev));
}

int checkResult(float* A, float* B, size_t size){
    for(int i = 0;i < size; i++){
        if(A[i] != B[i]){
            return i;
        }
    }
    return 0;
}
